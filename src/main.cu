#include "hip/hip_runtime.h"
#include "rt/test/debug.hh"
#include "rt/shape/sphere.hh"

#include <iostream>

// TODO: clean up CMakeLists.txt
// TODO: meer tests
// TODO: static constructors?
// TODO: fma for vectors?

__managed__ rt::intersection i;

__global__ void kernel(rt::sphere sphere, rt::ray3f ray) {
	intersect(sphere, ray, i);
}

int main() {
	rt::vec3f origin = rt::vector<rt::float_t>(1, 1, 1);
	rt::vec3f direction = normalize(-origin);
	rt::ray3f ray(origin, direction, 0);
	rt::sphere sphere(1);
	kernel<<<1, 1>>>(sphere, ray);
	hipDeviceSynchronize();
	std::cout << i << std::endl;
}
