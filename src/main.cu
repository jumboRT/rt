#include "hip/hip_runtime.h"
#include "rt/test/debug.hh"
#include "rt/shape/sphere.hh"

#include <iostream>
#include <random>
#include <tuple>

// TODO: clean up CMakeLists.txt
// TODO: meer tests

__managed__ rt::intersection i;

__global__ void kernel(rt::sphere sphere, rt::ray3f ray) {
	intersect(sphere, ray, i);
}

int main() {
	rt::vec3f origin = rt::vec3f(1, 1, 1);
	rt::vec3f direction = normalize(-origin);
	rt::ray3f ray(origin, direction, 0);
	rt::sphere sphere(1);
	kernel<<<1, 1>>>(sphere, ray);
	hipDeviceSynchronize();
	std::cout << i << std::endl;
}
