#include "hip/hip_runtime.h"
#include "rt/test/debug.hh"
#include "rt/shape/sphere.hh"

#include <iostream>

// TODO: clean up CMakeLists.txt
// TODO: move rt::swap to a different header
// TODO: rt::span const "begin" en "end"

__managed__ rt::intersection i;

__global__ void kernel(rt::sphere sphere, rt::ray3f ray) {
	sphere.intersect(ray, i);
}

int main() {
	rt::vec3f origin = rt::vector<rt::real>(1, 1, 1);
	rt::vec3f direction = rt::normalize(-origin);
	rt::ray3f ray(origin, direction, 0);
	rt::sphere sphere(1);
	kernel<<<1, 1>>>(sphere, ray);
	hipDeviceSynchronize();
	std::cout << i << std::endl;
}
